#include "hip/hip_runtime.h"
#include <pvkernel/core/general.h>
#include <pvkernel/cuda/common.h>
#include <pvparallelview/common.h>
#include <pvparallelview/PVBCICode.h>
#include <pvparallelview/PVHSVColor.h>
#include "bci_cuda.h"

#define NTHREADS_BLOCK 1024
#define SMEM_IMG_KB (4*4)
#define NBANDS_THREAD (SMEM_IMG_KB/4)

#define SMEM_NBCI (NTHREADS_BLOCK)

// From http://code.google.com/p/cudaraster/source/browse/trunk/src/cudaraster/cuda/Util.hpp?r=4
// See ptx_isa_3.0.pdf in CUDA SDK documentation for more information on prmt.b32
__device__ __inline__ unsigned int prmt(unsigned int a, unsigned int b, unsigned int c)
{
	unsigned int v;
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c));
	return v;
}

using PVParallelView::PVBCICode;

#define MASK_ZBUFFER 0x00FFFFFF

#pragma pack(push)
#pragma pack(4)
struct img_zbuffer_t
{
	union {
		uint32_t int_v;
		struct {
			uint8_t zbuffer[3];
			uint8_t hsv;
		} s;
	};
};
#pragma pack(pop)

__device__ __inline__ unsigned char zone2pos(unsigned char zone)
{
	const unsigned char a0 = zone&1;
	const unsigned char a1 = (zone&2)>>1;
	const unsigned char a2 = zone&4;

	return ((!(a0 ^ a1)) & (!a2)) |
	      (((a1 & (!a0)) | ((a2>>2) & a0)) << 1);
}

__device__ __inline__ unsigned char plus1mod3(unsigned char i)
{
	//return (i+1)%3;
	const unsigned char a0 = i&1;
	const unsigned char a1 = i&2;

	return (!i) | (((!a1) & a0)<<1);

}

__device__ __noinline__ unsigned int hsv2rgb(unsigned int hsv)
{
	// We have:
	// hsv defines actually only h which is divided in zone of 2**HSV_COLOR_NBITS_ZONE numbers.
	// Thus, we need to compute the zone, pos and mask, and then
	// v = {R, G, B}
	// v[pos] = x ^ mask
	// v[(pos+1)%3] = 0 ^ mask
	// v[(pos+2)%3] = 255 ^ mask
	
	unsigned char zone = (unsigned char) (hsv>>HSV_COLOR_NBITS_ZONE);
	unsigned char pos = zone2pos(zone);
	unsigned char mask = (zone&1)*0xFF;
	
	unsigned int pre_perm0, pre_perm1, pre_perm2,pre_perm,perm;
	pre_perm0 = (((hsv&HSV_COLOR_MASK_ZONE)*255)>>HSV_COLOR_NBITS_ZONE) ^ mask;
	pre_perm1 = mask;
	pre_perm2 = 0xFF ^ mask;
	asm("mov.u32 %0,{%1,%2,%3,%4};" : "=r"(pre_perm) : "r"(pre_perm0), "r"(pre_perm1), "r"(pre_perm2), "r"((unsigned int)0xFF));

	const unsigned int pos2 = plus1mod3(pos);
	perm = (1 << (pos2<<2)) | (2 << ((plus1mod3(pos2))<<2)) | (3 << 12);

	return prmt(pre_perm, pre_perm, perm);
}

__global__ void bcicode_raster(uint2* bci_codes, unsigned int n, unsigned int width, unsigned int* img_dst/*[width][IMAGE_HEIGHT]*/)
{
	__shared__ unsigned int shared_img[(SMEM_IMG_KB*1024)/sizeof(unsigned int)];
	__shared__ uint2 shared_bci[SMEM_NBCI];

	// The x coordinate of the band this thread is responsible of
	int band_x = threadIdx.x + blockIdx.x*blockDim.x;
	if (band_x >= width) {
		return;
	}

	const unsigned int bci_block_idx = blockIdx.x * blockIdx.y*gridDim.x;
	const unsigned int bci_size_grid = gridDim.x*gridDim.y*blockDim.x*blockDim.y;
	const unsigned int img_size_grid = blockDim.y*gridDim.y;
	unsigned int bci_thread_idx = threadIdx.x + threadIdx.y*blockDim.x;

	const float alpha = (float)(width-band_x)/(float)width;
	
	// First stage is to set shared memory
	for (int y = threadIdx.y; y < IMAGE_HEIGHT; y += img_size_grid) {
		shared_img[threadIdx.x + y*blockDim.x] = 0xFFFFFFFF;
	}

	__syncthreads();

	unsigned int idx_codes = bci_block_idx*(blockDim.x*blockDim.y) + bci_thread_idx;
	for (; idx_codes < n; idx_codes += bci_size_grid) {
		shared_bci[bci_thread_idx] = bci_codes[idx_codes];
		__syncthreads();

		unsigned int bci_read_thread_idx;
		for (unsigned int tx = 0; tx < blockDim.x; tx++) {
			bci_read_thread_idx = tx + threadIdx.y*blockDim.x;
			uint2 code0 = shared_bci[bci_read_thread_idx];
			code0.x >>= 8;
			float l0 = (float) (code0.y & 0x3ff);
			float r0 = (float) ((code0.y & 0xffc00)>>10);
			int pixel_y0 = (int) (r0 + ((l0-r0)*alpha) + 0.5f);
			unsigned int idx_shared_img0 = threadIdx.x + pixel_y0*blockDim.x;
			unsigned int cur_shared_p = shared_img[idx_shared_img0];
			unsigned int color0 = (code0.y & 0xff00000)<<4;
			if ((cur_shared_p & MASK_ZBUFFER) > code0.x) {
				shared_img[idx_shared_img0] = color0 | code0.x;
			}
		}
	}

	__syncthreads();

	
	// Final stage is to commit the shared image into the global image
	for (int y = threadIdx.y; y < IMAGE_HEIGHT; y += img_size_grid) {
		unsigned int pixel = shared_img[threadIdx.x + y*blockDim.x]>>24;
		if (pixel != 0xFF) {
			pixel = hsv2rgb(pixel);
		}
		else {
			pixel = 0xFFFFFFFF;
		}
		img_dst[band_x + y*width] = pixel;
	}
}

void show_codes_cuda(PVParallelView::PVBCICode* codes, uint32_t n, uint32_t width, uint32_t* img_dst/*[width][IMAGE_HEIGHT]*/)
{
	PVBCICode* device_codes;
	uint32_t* device_img;
	picviz_verify(sizeof(PVBCICode) == sizeof(uint64_t));

	picviz_verify_cuda(hipMalloc(&device_codes, n*sizeof(PVBCICode)));
	picviz_verify_cuda(hipMemcpy(device_codes, codes, n*sizeof(PVBCICode), hipMemcpyHostToDevice));

	size_t simg = width*IMAGE_HEIGHT*sizeof(uint32_t);
	picviz_verify_cuda(hipMalloc(&device_img, simg));
	picviz_verify_cuda(hipMemset(device_img, 0, simg));
	
	hipEvent_t start,end;
	picviz_verify_cuda(hipEventCreate(&start));
	picviz_verify_cuda(hipEventCreate(&end));

	// Compute number of threads per block
	//int nthreads_x = picviz_min(width, PVCuda::get_shared_mem_size()/(IMAGE_HEIGHT*sizeof(img_zbuffer_t)));
	int nthreads_x = (picviz_min(width, (SMEM_IMG_KB*1024)/(IMAGE_HEIGHT*sizeof(img_zbuffer_t))));
	int nthreads_y = NTHREADS_BLOCK/nthreads_x;
	picviz_verify(nthreads_x*nthreads_y <= NTHREADS_BLOCK);
	PVLOG_INFO("Number threads per block: %d x %d\n", nthreads_x, nthreads_y);

	// Compute number of blocks
	int nblocks = PVCuda::get_number_blocks();
	int nblocks_x = (width+nthreads_x-1)/nthreads_x;
	int nblocks_y = 1;
	picviz_verify(nblocks_y > 0);
	PVLOG_INFO("Number of blocks: %d x %d\n", nblocks_x, nblocks_y);

	//int shared_size = nthreads_x*IMAGE_HEIGHT*sizeof(img_zbuffer_t);

	picviz_verify_cuda(hipEventRecord(start, 0));
	bcicode_raster<<<dim3(nblocks_x,nblocks_y),dim3(nthreads_x, nthreads_y)>>>((uint2*) device_codes, n, width, device_img);
	picviz_verify_cuda_kernel();
	picviz_verify_cuda(hipEventRecord(end, 0));
	picviz_verify_cuda(hipEventSynchronize(end));

	picviz_verify_cuda(hipMemcpy(img_dst, device_img, simg, hipMemcpyDeviceToHost));

	picviz_verify_cuda(hipFree(device_codes));
	picviz_verify_cuda(hipFree(device_img));
	
	float time = 0;
	picviz_verify_cuda(hipEventElapsedTime(&time, start, end));

	fprintf(stdout, "CUDA kernel time: %0.4f ms, BW: %0.4f MB/s\n", time, (double)(n*sizeof(PVBCICode))/(double)((time/1000.0)*1024.0*1024.0));
}
