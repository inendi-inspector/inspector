#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @copyright (C) Picviz Labs 2010-March 2015
 * @copyright (C) ESI Group INENDI April 2015-2015
 */

#include "selection_kernels.h"

__global__ void picviz_selection_AB2C_or_k(uint32_t *da, uint32_t *db, uint32_t *dc)
{
	int chunk_index = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	
	for (i=chunk_index*100;i < (chunk_index+1)*100;++i) {
//	    da[i] = 0xff;
	    dc[i] = da[i] | db[i];
 	}
}
