#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @copyright (C) Picviz Labs 2010-March 2015
 * @copyright (C) ESI Group INENDI April 2015-2015
 */

#include <pvkernel/core/PVHSVColor.h>
#include <pvkernel/cuda/common.h>

#include <pvparallelview/common.h>
#include <pvparallelview/PVBCICode.h>
#include <pvparallelview/cuda/bci_cuda.h>

#include <cassert>
#include <algorithm>
#include <iostream>
#include <stdio.h>

#define NTHREADS_BLOCK 512
#define SMEM_IMG_KB (4*4)

// From http://code.google.com/p/cudaraster/source/browse/trunk/src/cudaraster/cuda/Util.hpp?r=4
// See ptx_isa_3.0.pdf in CUDA SDK documentation for more information on prmt.b32
__device__ __inline__ unsigned int prmt(unsigned int a, unsigned int b, unsigned int c)
{
	unsigned int v;
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c));
	return v;
}

using PVParallelView::PVBCICode;

//#define MASK_ZBUFFER 0x00FFFFFF
#define MASK_ZBUFFER 0xFFFFFF00
#define MASK_COLOR   0x000000FF
#define BCI_MASK_TYPE ((1<<2)-1)

#pragma pack(push)
#pragma pack(4)
struct img_zbuffer_t
{
	union {
		uint32_t int_v;
		struct {
			uint8_t zbuffer[3];
			uint8_t hsv;
		} s;
	};
};
#pragma pack(pop)

__device__ __inline__ unsigned char zone2pos(unsigned char zone)
{
	const unsigned char a0 = zone&1;
	const unsigned char a1 = (zone&2)>>1;
	const unsigned char a2 = zone&4;

	return ((!(a0 ^ a1)) & (!a2)) |
	      (((a1 & (!a0)) | ((a2>>2) & a0)) << 1);
}

__device__ __inline__ unsigned char plus1mod3(unsigned char i)
{
	//return (i+1)%3;
	const unsigned char a0 = i&1;
	const unsigned char a1 = i&2;

	return (!i) | (((!a1) & a0)<<1);

}

__device__ __noinline__ unsigned int hsv2rgb(unsigned int hsv)
{
	// We have:
	// hsv defines actually only h which is divided in zone of 2**HSV_COLOR_NBITS_ZONE numbers.
	// Thus, we need to compute the zone, pos and mask, and then
	// v = {R, G, B}
	// v[pos] = x ^ mask
	// v[(pos+1)%3] = 0 ^ mask
	// v[(pos+2)%3] = 255 ^ mask
	
	if (hsv == HSV_COLOR_WHITE) {
		return 0xFFFFFFFF; // Special value for white
	}
	if (hsv == HSV_COLOR_BLACK) {
		return 0xFF000000; // Special value for black
	}

	unsigned char zone = (unsigned char) (hsv>>HSV_COLOR_NBITS_ZONE);
	unsigned char pos = zone2pos(zone);
	unsigned char mask = (zone&1)*0xFF;
	
	unsigned int pre_perm0, pre_perm1, pre_perm2,pre_perm,perm;
	pre_perm0 = (((hsv&HSV_COLOR_MASK_ZONE)*255)>>HSV_COLOR_NBITS_ZONE) ^ mask;
	pre_perm1 = mask;
	pre_perm2 = 0xFF ^ mask;
	asm("mov.u32 %0,{%1,%2,%3,%4};" : "=r"(pre_perm) : "r"(pre_perm0), "r"(pre_perm1), "r"(pre_perm2), "r"((unsigned int)0xFF));

	const unsigned int pos2 = plus1mod3(pos);
	perm = (1 << (pos2<<2)) | (2 << ((plus1mod3(pos2))<<2)) | (3 << 12);

	return prmt(pre_perm, pre_perm, perm);
}

template <size_t Bbits, bool width_reverse>
__global__ void bcicode_raster_unroll2(uint2* bci_codes, unsigned int n, unsigned int width,  unsigned int* img_dst, unsigned int img_width, unsigned int img_x_start, const float zoom_y)
{
	// shared_size = blockDim.x*IMAGE_HEIGHT*sizeof(img_zbuffer_t)
	__shared__ unsigned int shared_img[(SMEM_IMG_KB*1024)/sizeof(unsigned int)];

	// The x coordinate of the band this thread is responsible of
	int band_x = threadIdx.x + blockIdx.x*blockDim.x;
	if (band_x >= width) {
		return;
	}

	// Do this division once and for all
	const float alpha0 = (float)(width-band_x)/(float)width;
	const float alpha1 = (float)(width-(band_x+1))/(float)width;
	const unsigned int y_start = threadIdx.y + blockIdx.y*blockDim.y;
	const unsigned int size_grid = blockDim.y*gridDim.y;

	// First stage is to set shared memory
	for (int y = threadIdx.y; y < PVParallelView::constants<Bbits>::image_height; y += blockDim.y) {
		shared_img[threadIdx.x + y*blockDim.x] = 0xFFFFFFFF;
	}

#if 0
	const unsigned int size_grid2 = size_grid<<1;
	const unsigned int n_end = (n/(size_grid2))*(size_grid2);
#endif

	__syncthreads();

	unsigned int idx_codes = y_start;
#if 0
	for (; idx_codes < n_end; idx_codes += size_grid2) {
		uint2 code0 = bci_codes[idx_codes];
		uint2 code1 = bci_codes[idx_codes+size_grid];
		uint2 code2 = bci_codes[idx_codes+size_grid*2];
		uint2 code3 = bci_codes[idx_codes+size_grid*3];

		/*if (threadIdx.x == 0 && threadIdx.y == 0) {
			i++;
		}*/
		
		// For information:
		// struct PVBCICode
		// {
		//	typedef PVCore::PVAlignedAllocator<PVBCICode, 16> allocator;
		//	union {
		//		uint64_t int_v;
		//		struct {
		//			uint32_t idx;
		//			uint32_t l: 10;
		//			uint32_t r: 10;
		//			uint32_t color: 9;
		//			uint32_t __reserved: 1;
		//		} s;
		//	};
		// }

		// 24-bit z-buffer
		code0.x >>= 8; code1.x >>= 8; code2.x >>= 8; code3.x >>= 8;

		// Get l, r and color
		const float l0 = (float) (code0.y & 0x3ff);
		const float r0 = (float) ((code0.y & 0xffc00)>>10);
		const float l1 = (float) (code1.y & 0x3ff);
		const float r1 = (float) ((code1.y & 0xffc00)>>10);
		const float l2 = (float) (code2.y & 0x3ff);
		const float r2 = (float) ((code2.y & 0xffc00)>>10);
		const float l3 = (float) (code3.y & 0x3ff);
		const float r3 = (float) ((code3.y & 0xffc00)>>10);

		// Compute the y coordinate for band_x
		const int pixel_y0 = (int) (r0 + ((l0-r0)*alpha) + 0.5f);
		const int pixel_y1 = (int) (r1 + ((l1-r1)*alpha) + 0.5f);
		const int pixel_y2 = (int) (r2 + ((l2-r2)*alpha) + 0.5f);
		const int pixel_y3 = (int) (r3 + ((l3-r3)*alpha) + 0.5f);
		unsigned int idx_shared_img0 = threadIdx.x + pixel_y0*blockDim.x;
		unsigned int idx_shared_img1 = threadIdx.x + pixel_y1*blockDim.x;
		unsigned int idx_shared_img2 = threadIdx.x + pixel_y2*blockDim.x;
		unsigned int idx_shared_img3 = threadIdx.x + pixel_y3*blockDim.x;
		const unsigned int color0 = (code0.y & 0xff00000)<<4;
		const unsigned int color1 = (code1.y & 0xff00000)<<4;
		const unsigned int color2 = (code2.y & 0xff00000)<<4;
		const unsigned int color3 = (code3.y & 0xff00000)<<4;

		// Set shared_img
		unsigned int cur_shared_p0 = shared_img[idx_shared_img0] & MASK_ZBUFFER;
		if (cur_shared_p0 > code0.x) {
			shared_img[idx_shared_img0] = color0 | code0.x;
		}
		unsigned int cur_shared_p1 = shared_img[idx_shared_img1] & MASK_ZBUFFER;
		if (cur_shared_p1 > code1.x) {
			shared_img[idx_shared_img1] = color1 | code1.x;
		}
		unsigned int cur_shared_p2 = shared_img[idx_shared_img2] & MASK_ZBUFFER;
		if (cur_shared_p2 > code2.x) {
			shared_img[idx_shared_img2] = color2 | code2.x;
		}
		unsigned int cur_shared_p3 = shared_img[idx_shared_img3] & MASK_ZBUFFER;
		if (cur_shared_p3 > code3.x) {
			shared_img[idx_shared_img3] = color3 | code3.x;
		}
		__syncthreads();
	}
	for (; idx_codes < n_end; idx_codes += size_grid2) {
		uint2 code0 = bci_codes[idx_codes];
		uint2 code1 = bci_codes[idx_codes+size_grid];
		
		// For information:
		// struct PVBCICode
		// {
		//	typedef PVCore::PVAlignedAllocator<PVBCICode, 16> allocator;
		//	union {
		//		uint64_t int_v;
		//		struct {
		//			uint32_t idx;
		//			uint32_t l: 10;
		//			uint32_t r: 10;
		//			uint32_t color: 9;
		//			uint32_t __reserved: 1;
		//		} s;
		//	};
		// }

		// 24-bit z-buffer
		code0.x &= MASK_ZBUFFER; code1.x &= MASK_ZBUFFER;

		// Get l, r and color
		const float l0 = (float) (code0.y & PVParallelView::constants<Bbits>::mask_int_ycoord);
		const float r0 = (float) ((code0.y >> Bbits) & PVParallelView::constants<Bbits>::mask_int_ycoord);
		const float l1 = (float) (code1.y & PVParallelView::constants<Bbits>::mask_int_ycoord);
		const float r1 = (float) ((code1.y >> Bbits) & PVParallelView::constants<Bbits>::mask_int_ycoord);

		// Compute the y coordinate for band_x
		int pixel_y00 = (int) (((r0 + ((l0-r0)*alpha0)) * zoom_y) + 0.5f);
		int pixel_y01 = (int) (((r0 + ((l0-r0)*alpha1)) * zoom_y) + 0.5f);
		if (pixel_y00 > pixel_y01) {
			const int tmp = pixel_y00;
			pixel_y00 = pixel_y01;
			pixel_y01 = tmp;
		}
		int pixel_y10 = (int) (((r1 + ((l1-r1)*alpha0)) * zoom_y) + 0.5f);
		int pixel_y11 = (int) (((r1 + ((l1-r1)*alpha1)) * zoom_y) + 0.5f);
		if (pixel_y10 > pixel_y11) {
			const int tmp = pixel_y10;
			pixel_y10 = pixel_y11;
			pixel_y11 = tmp;
		}
		const unsigned int shared_v0 = ((code0.y >> 2*Bbits) & 0xff) | code0.x;
		const unsigned int shared_v1 = ((code1.y >> 2*Bbits) & 0xff) | code1.x;

		atomicMin(&shared_img[threadIdx.x + pixel_y00*blockDim.x], shared_v0);
		atomicMin(&shared_img[threadIdx.x + pixel_y10*blockDim.x], shared_v1);

		for (int pixel_y0 = pixel_y00+1; pixel_y0 < pixel_y01; pixel_y0++) {
			atomicMin(&shared_img[threadIdx.x + pixel_y0*blockDim.x], shared_v0);
		}
		for (int pixel_y1 = pixel_y10+1; pixel_y1 < pixel_y11; pixel_y1++) {
			atomicMin(&shared_img[threadIdx.x + pixel_y1*blockDim.x], shared_v1);
		}
	}
#endif
	for (; idx_codes < n; idx_codes += size_grid) {
		uint2 code0 = bci_codes[idx_codes];
		code0.x &= MASK_ZBUFFER;
		const float l0 = (float) (code0.y & PVParallelView::constants<Bbits>::mask_int_ycoord);
		const int r0i = (code0.y >> Bbits) & PVParallelView::constants<Bbits>::mask_int_ycoord;
		const int type = (code0.y >> ((2*Bbits) + 8)) & BCI_MASK_TYPE;
		int pixel_y00;
		int pixel_y01;
		if (type == PVBCICode<Bbits>::STRAIGHT) {
			const float r0 = (float) r0i;
			pixel_y00 = (int) (((r0 + ((l0-r0)*alpha0)) * zoom_y) + 0.5f);
			pixel_y01 = (int) (((r0 + ((l0-r0)*alpha1)) * zoom_y) + 0.5f);
		}
		else {
			// TODO: optimise this !!
			// 'r0i' is x
			if (band_x > r0i) {
				// This is out of our drawing scope !
				continue;
			}
			const float r0 = (float) r0i;
			if (type == PVBCICode<Bbits>::UP) {
				const float alpha_x = l0/r0;
				pixel_y00 = (int) (((l0-(alpha_x*(float)band_x))*zoom_y) + 0.5f);
				if (band_x == r0i) {
					pixel_y01 = pixel_y00;
				}
				else {
					pixel_y01 = (int) (((l0-(alpha_x*(float)(band_x+1)))*zoom_y) + 0.5f);
				}
			}
			else {
				const float alpha_x = ((float)PVParallelView::constants<Bbits>::mask_int_ycoord-l0)/r0;
				pixel_y00 = (int) (((l0+(alpha_x*(float)band_x))*zoom_y) + 0.5f);
				if (band_x == r0i) {
					pixel_y01 = pixel_y00;
				}
				else {
					pixel_y01 = (int) (((l0+(alpha_x*(float)(band_x+1)))*zoom_y) + 0.5f);
				}
			}
		}

		if (pixel_y00 > pixel_y01) {
			const int tmp = pixel_y00;
			pixel_y00 = pixel_y01;
			pixel_y01 = tmp;
		}

		const unsigned int color0 = (code0.y >> 2*Bbits) & 0xff;
		if (color0 == HSV_COLOR_BLACK) { // Used for zombie events, so their index is the highest possible (behind everyone)
			code0.x = MASK_ZBUFFER;
		}
		const unsigned int shared_v = color0 | code0.x;
		atomicMin(&shared_img[threadIdx.x + pixel_y00*blockDim.x], shared_v);
		for (int pixel_y0 = pixel_y00+1; pixel_y0 < pixel_y01; pixel_y0++) {
			atomicMin(&shared_img[threadIdx.x + pixel_y0*blockDim.x], shared_v);
		}
	}
	
	band_x += img_x_start;
	if (width_reverse) {
		band_x = img_width-band_x-1;
	}
	__syncthreads();
	// Final stage is to commit the shared image into the global image
	for (int y = threadIdx.y; y < PVParallelView::constants<Bbits>::image_height; y += blockDim.y) {
		const unsigned int pixel_shared = shared_img[threadIdx.x + y*blockDim.x];
		unsigned int pixel;
		if (pixel_shared != 0xFFFFFFFF) {
			pixel = hsv2rgb(pixel_shared & MASK_COLOR);
		}
		else {
			pixel = 0x00000000; // Transparent background
		}
		img_dst[band_x + y*img_width] = pixel;
	}
}

template <size_t Bbits>
static inline int get_nthread_x_from_width(int width)
{
	return std::min(width, (int)((SMEM_IMG_KB*1024)/(PVParallelView::constants<Bbits>::image_height*sizeof(img_zbuffer_t))));
}

template <size_t Bbits, bool reverse>
static void show_codes_cuda(PVParallelView::PVBCICode<Bbits>* device_codes, uint32_t n, uint32_t width, uint32_t* device_img, uint32_t img_width, uint32_t x_start, const float zoom_y, hipStream_t stream)
{
	if (zoom_y == 0) {
		return;
	}
	assert((zoom_y > 0) && (zoom_y <= 1.0f));
	// Compute number of threads per block
	int nthreads_x = get_nthread_x_from_width<Bbits>(width);
	int nthreads_y = NTHREADS_BLOCK/nthreads_x;
	assert(nthreads_x*nthreads_y <= NTHREADS_BLOCK);

	// Compute number of blocks
	int nblocks = PVCuda::get_number_blocks();
	int nblocks_x = (width+nthreads_x-1)/nthreads_x;
	int nblocks_y = 1;

	// Launch CUDA kernel!
	bcicode_raster_unroll2<Bbits, reverse><<<dim3(nblocks_x,nblocks_y),dim3(nthreads_x, nthreads_y), 0, stream>>>((uint2*) device_codes, n, width, device_img, img_width, x_start, zoom_y);
}

template <size_t Bbits, bool reverse>
static float show_and_perf_codes_cuda(PVParallelView::PVBCICode<Bbits>* device_codes, uint32_t n, uint32_t width, uint32_t* device_img, uint32_t img_width, uint32_t x_start, const float zoom_y, hipStream_t stream, double* kernel_bw)
{
	// WARNING!
	// This will imply a synchronous call to the CUDA kernel, thus must not be used
	// inside the Picviz rendering pipeline!!
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreateWithFlags(&stop, hipEventBlockingSync);

	hipEventRecord(start, stream);
	show_codes_cuda<Bbits, reverse>(device_codes, n, width, device_img, img_width, x_start, zoom_y, stream);
	picviz_verify_cuda_kernel();
	hipEventRecord(stop, stream);
	hipEventSynchronize(stop);

	float time = 0.0f;
	hipEventElapsedTime(&time, start, stop);
	if (kernel_bw) {
		*kernel_bw = (double)(n*sizeof(PVBCICode<Bbits>))/(double)((time/1000.0)*1024.0*1024.0);
	}

	return time;
}

void show_codes_cuda10(PVParallelView::PVBCICode<10>* device_codes, uint32_t n, uint32_t width, uint32_t* device_img, uint32_t img_width, uint32_t x_start, const float zoom_y, hipStream_t stream)
{
	show_codes_cuda<10, false>(device_codes, n, width, device_img, img_width, x_start, zoom_y, stream);
}

void show_codes_cuda11(PVParallelView::PVBCICode<11>* device_codes, uint32_t n, uint32_t width, uint32_t* device_img, uint32_t img_width, uint32_t x_start, const float zoom_y, hipStream_t stream)
{
	show_codes_cuda<11, false>(device_codes, n, width, device_img, img_width, x_start, zoom_y, stream);
}

void show_codes_cuda11_reverse(PVParallelView::PVBCICode<11>* device_codes, uint32_t n, uint32_t width, uint32_t* device_img, uint32_t img_width, uint32_t x_start, const float zoom_y, hipStream_t stream)
{
	show_codes_cuda<11, true>(device_codes, n, width, device_img, img_width, x_start, zoom_y, stream);
}

float show_and_perf_codes_cuda10(PVParallelView::PVBCICode<10>* device_codes, uint32_t n, uint32_t width, uint32_t* device_img, uint32_t img_width, uint32_t x_start, const float zoom_y, hipStream_t stream, double* bw)
{
	return show_and_perf_codes_cuda<10, false>(device_codes, n, width, device_img, img_width, x_start, zoom_y, stream, bw);
}

float show_and_perf_codes_cuda11(PVParallelView::PVBCICode<11>* device_codes, uint32_t n, uint32_t width, uint32_t* device_img, uint32_t img_width, uint32_t x_start, const float zoom_y, hipStream_t stream, double* bw)
{
	return show_and_perf_codes_cuda<11, false>(device_codes, n, width, device_img, img_width, x_start, zoom_y, stream, bw);
}
