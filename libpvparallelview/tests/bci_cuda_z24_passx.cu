#include "hip/hip_runtime.h"
/**
 * @file
 *
 * @copyright (C) Picviz Labs 2010-March 2015
 * @copyright (C) ESI Group INENDI April 2015-2015
 */

#include <pvkernel/core/general.h>
#include <pvkernel/cuda/common.h>
#include <pvparallelview/common.h>
#include <pvparallelview/PVBCICode.h>
#include <pvkernel/core/PVHSVColor.h>
#include "bci_cuda.h"

#define NTHREADS_BLOCK 1024
#define SMEM_IMG_KB (12*4)
#define NCODES_SHARED 512
#define NBANDS_SHARED (SMEM_IMG_KB/4)
#define NPASS_SHARED 2

// From http://code.google.com/p/cudaraster/source/browse/trunk/src/cudaraster/cuda/Util.hpp?r=4
// See ptx_isa_3.0.pdf in CUDA SDK documentation for more information on prmt.b32
__device__ __inline__ unsigned int prmt(unsigned int a, unsigned int b, unsigned int c)
{
	unsigned int v;
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c));
	return v;
}

using PVParallelView::PVBCICode;

#define MASK_ZBUFFER 0x00FFFFFF

#pragma pack(push)
#pragma pack(4)
struct img_zbuffer_t
{
	union {
		uint32_t int_v;
		struct {
			uint8_t zbuffer[3];
			uint8_t hsv;
		} s;
	};
};
#pragma pack(pop)

__device__ __inline__ unsigned char zone2pos(unsigned char zone)
{
	const unsigned char a0 = zone&1;
	const unsigned char a1 = (zone&2)>>1;
	const unsigned char a2 = zone&4;

	return ((!(a0 ^ a1)) & (!a2)) |
	      (((a1 & (!a0)) | ((a2>>2) & a0)) << 1);
}

__device__ __inline__ unsigned char plus1mod3(unsigned char i)
{
	//return (i+1)%3;
	const unsigned char a0 = i&1;
	const unsigned char a1 = i&2;

	return (!i) | (((!a1) & a0)<<1);

}

__device__ __noinline__ unsigned int hsv2rgb(unsigned int hsv)
{
	// We have:
	// hsv defines actually only h which is divided in zone of 2**HSV_COLOR_NBITS_ZONE numbers.
	// Thus, we need to compute the zone, pos and mask, and then
	// v = {R, G, B}
	// v[pos] = x ^ mask
	// v[(pos+1)%3] = 0 ^ mask
	// v[(pos+2)%3] = 255 ^ mask
	
	unsigned char zone = (unsigned char) (hsv>>HSV_COLOR_NBITS_ZONE);
	unsigned char pos = zone2pos(zone);
	unsigned char mask = (zone&1)*0xFF;
	
	unsigned int pre_perm0, pre_perm1, pre_perm2,pre_perm,perm;
	pre_perm0 = (((hsv&HSV_COLOR_MASK_ZONE)*255)>>HSV_COLOR_NBITS_ZONE) ^ mask;
	pre_perm1 = mask;
	pre_perm2 = 0xFF ^ mask;
	asm("mov.u32 %0,{%1,%2,%3,%4};" : "=r"(pre_perm) : "r"(pre_perm0), "r"(pre_perm1), "r"(pre_perm2), "r"((unsigned int)0xFF));

	const unsigned int pos2 = plus1mod3(pos);
	perm = (1 << (pos2<<2)) | (2 << ((plus1mod3(pos2))<<2)) | (3 << 12);

	return prmt(pre_perm, pre_perm, perm);
}

__global__ void bcicode_raster_unroll2(uint2* bci_codes, unsigned int n, unsigned int width, unsigned int* img_dst/*[width][IMAGE_HEIGHT]*/)
{
	__shared__ unsigned int shared_img[(SMEM_IMG_KB*1024)/sizeof(unsigned int)];

	const unsigned int y_start = threadIdx.y + blockIdx.y*blockDim.y;
	const unsigned int size_grid = blockDim.y*gridDim.y;

	// First stage is to set shared memory
	const unsigned int block_dim_bands = blockDim.x*NBANDS_THREAD;
#if 0
	for (int y = y_start; y < IMAGE_HEIGHT; y += size_grid) {
		for (int x = 0; x < NBANDS_THREAD; x++) {
			shared_img[x + y*block_dim_bands] = 0xFFFFFFFF;
		}
	}
#endif

	const unsigned int size_grid2 = size_grid<<1;
	const unsigned int n_end = (n/(size_grid2))*(size_grid2);

	__syncthreads();

	unsigned int idx_codes = y_start;
	for (; idx_codes < n; idx_codes += size_grid) {
		uint2 code0 = bci_codes[idx_codes];
		code0.x >>= 8;
		float l0 = (float) (code0.y & 0x3ff);
		float r0 = (float) ((code0.y & 0xffc00)>>10);
		unsigned int color0 = (code0.y & 0xff00000)<<4;

		for (int shared_pass = 0; shared_pass < NPASS_SHARED; shared_pass++) {
			const int x_start = blockIdx.x*block_dim_bands + ;
			int nbands = NBANDS_THREAD;
			if (x_start + NBANDS_THREAD >= width) {
				nbands = width-x_start;
			}
			for (int x = 0; x < nbands; x++) {
				int band_x = x_start+x;
				const float alpha = (float)(width-band_x)/(float)width;
				int pixel_y0 = (int) (r0 + ((l0-r0)*alpha) + 0.5f);
				unsigned int idx_shared_img0 = x + pixel_y0*block_dim_bands;
				unsigned int cur_shared_p = shared_img[idx_shared_img0];
				if ((cur_shared_p & MASK_ZBUFFER) > code0.x) {
					shared_img[idx_shared_img0] = color0 | code0.x;
				}
			}
	}

	__syncthreads();

	
	// Final stage is to commit the shared image into the global image
	for (int y = y_start; y < IMAGE_HEIGHT; y += size_grid) {
		for (int x = 0; x < nbands; x++) {
			int band_x = x_start+x;
			unsigned int pixel = shared_img[x + y*block_dim_bands]>>24;
			if (pixel != 0xFF) {
				pixel = hsv2rgb(pixel);
			}
			else {
				pixel = 0xFFFFFFFF;
			}
			img_dst[band_x + y*width] = pixel;
		}
	}
}

void show_codes_cuda(PVParallelView::PVBCICode<>* codes, uint32_t n, uint32_t width, uint32_t* img_dst/*[width][IMAGE_HEIGHT]*/)
{
	PVBCICode<>* device_codes;
	uint32_t* device_img;
	inendi_verify(sizeof(PVBCICode<>) == sizeof(uint64_t));

	inendi_verify_cuda(hipMalloc(&device_codes, n*sizeof(PVBCICode<>)));
	inendi_verify_cuda(hipMemcpy(device_codes, codes, n*sizeof(PVBCICode<>), hipMemcpyHostToDevice));

	size_t simg = width*IMAGE_HEIGHT*sizeof(uint32_t);
	inendi_verify_cuda(hipMalloc(&device_img, simg));
	inendi_verify_cuda(hipMemset(device_img, 0xFF, simg));
	
	hipEvent_t start,end;
	inendi_verify_cuda(hipEventCreate(&start));
	inendi_verify_cuda(hipEventCreate(&end));

	// Compute number of threads per block
	//int nthreads_x = inendi_min(width, PVCuda::get_shared_mem_size()/(IMAGE_HEIGHT*sizeof(img_zbuffer_t)));
	//int nthreads_x = (inendi_min(width, (SMEM_IMG_KB*1024)/(IMAGE_HEIGHT*sizeof(img_zbuffer_t))));
	int nthreads_x = 1;
	int nthreads_y = NTHREADS_BLOCK/nthreads_x;
	inendi_verify(nthreads_x*nthreads_y <= NTHREADS_BLOCK);
	PVLOG_INFO("Number threads per block: %d x %d\n", nthreads_x, nthreads_y);

	// Compute number of blocks
	int nblocks = PVCuda::get_number_blocks();
	int nblocks_x = ((width+nthreads_x*NBANDS_THREAD-1)/(nthreads_x*NBANDS_THREAD));
	int nblocks_y = 2;
	inendi_verify(nblocks_y > 0);
	PVLOG_INFO("Number of blocks: %d x %d\n", nblocks_x, nblocks_y);

	//int shared_size = nthreads_x*IMAGE_HEIGHT*sizeof(img_zbuffer_t);

	inendi_verify_cuda(hipEventRecord(start, 0));
	bcicode_raster_unroll2<<<dim3(nblocks_x,nblocks_y),dim3(nthreads_x, nthreads_y)>>>((uint2*) device_codes, n, width, device_img);
	inendi_verify_cuda_kernel();
	inendi_verify_cuda(hipEventRecord(end, 0));
	inendi_verify_cuda(hipEventSynchronize(end));

	inendi_verify_cuda(hipMemcpy(img_dst, device_img, simg, hipMemcpyDeviceToHost));

	inendi_verify_cuda(hipFree(device_codes));
	inendi_verify_cuda(hipFree(device_img));
	
	float time = 0;
	inendi_verify_cuda(hipEventElapsedTime(&time, start, end));

	fprintf(stderr, "CUDA kernel time: %0.4f ms, BW: %0.4f MB/s\n", time, (double)(n*sizeof(PVBCICode<>))/(double)((time/1000.0)*1024.0*1024.0));
}
